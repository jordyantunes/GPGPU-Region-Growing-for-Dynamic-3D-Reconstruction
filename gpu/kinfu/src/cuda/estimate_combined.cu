#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 * 
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

//#include <pcl/gpu/utils/device/block.hpp>
//#include <pcl/gpu/utils/device/funcattrib.hpp>
#include "device.hpp"

#define RANGA_MODIFICATION_DEPTHWEIGHT_CUTOFF 1
//#define RANGA_MODIFICATION_DEPTHWEIGHT_CURRENT_FRAME 1

namespace pcl
{
  namespace device
  {
    typedef double float_type;

    template<int CTA_SIZE_, typename T>
    static __device__ __forceinline__ void reduce(volatile T* buffer)
    {
      int tid = Block::flattenedThreadId();
      T val =  buffer[tid];

      if (CTA_SIZE_ >= 1024) { if (tid < 512) buffer[tid] = val = val + buffer[tid + 512]; __syncthreads(); }
      if (CTA_SIZE_ >=  512) { if (tid < 256) buffer[tid] = val = val + buffer[tid + 256]; __syncthreads(); }
      if (CTA_SIZE_ >=  256) { if (tid < 128) buffer[tid] = val = val + buffer[tid + 128]; __syncthreads(); }
      if (CTA_SIZE_ >=  128) { if (tid <  64) buffer[tid] = val = val + buffer[tid +  64]; __syncthreads(); }

      if (tid < 32)
      {
        if (CTA_SIZE_ >=   64) { buffer[tid] = val = val + buffer[tid +  32]; }
        if (CTA_SIZE_ >=   32) { buffer[tid] = val = val + buffer[tid +  16]; }
        if (CTA_SIZE_ >=   16) { buffer[tid] = val = val + buffer[tid +   8]; }
        if (CTA_SIZE_ >=    8) { buffer[tid] = val = val + buffer[tid +   4]; }
        if (CTA_SIZE_ >=    4) { buffer[tid] = val = val + buffer[tid +   2]; }
        if (CTA_SIZE_ >=    2) { buffer[tid] = val = val + buffer[tid +   1]; }
      }
    }

    struct Combined
    {
      enum
      {
        CTA_SIZE_X = 32,
        CTA_SIZE_Y = 8,
        CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y
      };


      Mat33 Rcurr;
      float3 tcurr;

      PtrStep<float> vmap_curr;
      PtrStep<float> nmap_curr;

      Mat33 Rprev_inv;
      float3 tprev;

      Intr intr;

      PtrStep<float> vmap_g_prev;
      PtrStep<float> nmap_g_prev;

      float distThres;
      float angleThres;

      int cols;
      int rows;

      mutable PtrStep<float_type> gbuf;

      __device__ __forceinline__ bool
      search (int x, int y, float3& n, float3& d, float3& s) const
      {
        float3 ncurr;
        ncurr.x = nmap_curr.ptr (y)[x];

        if (isnan (ncurr.x))
          return (false);

        float3 vcurr;
        vcurr.x = vmap_curr.ptr (y       )[x];
        vcurr.y = vmap_curr.ptr (y + rows)[x];
        vcurr.z = vmap_curr.ptr (y + 2 * rows)[x];

        float3 vcurr_g = Rcurr * vcurr + tcurr;

        float3 vcurr_cp = Rprev_inv * (vcurr_g - tprev);         // prev camera coo space

        int2 ukr;         //projection
        ukr.x = __float2int_rn (vcurr_cp.x * intr.fx / vcurr_cp.z + intr.cx);      //4
        ukr.y = __float2int_rn (vcurr_cp.y * intr.fy / vcurr_cp.z + intr.cy);                      //4

        if (ukr.x < 0 || ukr.y < 0 || ukr.x >= cols || ukr.y >= rows || vcurr_cp.z < 0)
          return (false);

        float3 nprev_g;
        nprev_g.x = nmap_g_prev.ptr (ukr.y)[ukr.x];

        if (isnan (nprev_g.x))
          return (false);

        float3 vprev_g;
        vprev_g.x = vmap_g_prev.ptr (ukr.y       )[ukr.x];
        vprev_g.y = vmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        vprev_g.z = vmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float dist = norm (vprev_g - vcurr_g);
        if (dist > distThres)
          return (false);

        ncurr.y = nmap_curr.ptr (y + rows)[x];
        ncurr.z = nmap_curr.ptr (y + 2 * rows)[x];

        float3 ncurr_g = Rcurr * ncurr;

        nprev_g.y = nmap_g_prev.ptr (ukr.y + rows)[ukr.x];
        nprev_g.z = nmap_g_prev.ptr (ukr.y + 2 * rows)[ukr.x];

        float sine = norm (cross (ncurr_g, nprev_g));

        if (sine >= angleThres)
          return (false);
        n = nprev_g;
        d = vprev_g;
        s = vcurr_g;
        return (true);
      }

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        float3 n, d, s, temp_1, temp_2;
        bool found_coresp = false;
		float depth = 0.f, min_depth = 0.8, max_depth = 4, weight = 1, step = 1;
		 float new_min_depth = 0;
		 float mind = 0.8, maxd = 4;

        if (x < cols && y < rows)
          found_coresp = search (x, y, n, d, s);

        float row[7];

        if (found_coresp)
        {
			depth = vmap_curr.ptr (y + 2 * rows)[x];

			if ( depth >= 0.8 && depth <= max_depth )
			{
#if RANGA_MODIFICATION_DEPTHWEIGHT_CUTOFF
				step = ((1/(min_depth*min_depth)) - (1/(max_depth*max_depth)));
				weight =  (((1/(depth*depth)) - (1/(max_depth*max_depth))) / step);
				weight = fabs(sqrt(weight));

				if(weight < 0.25)
					weight = 0.25;
#elif RANGA_MODIFICATION_DEPTHWEIGHT_CURRENT_FRAME
				int less_then_1500 = intr.number_less_than_1000 + intr.number_less_than_1500;
				int less_then_2000 = less_then_1500 + intr.number_less_than_2000;
				int less_then_2500 = less_then_2000 + intr.number_less_than_2500;
				int less_then_3000 = less_then_2500 + intr.number_less_than_3000;
				int less_then_3500 = less_then_3000 + intr.number_less_than_3500;
				int less_then_4000 = less_then_3500 + intr.number_less_than_4000;
				int disable_weights = 0;

				if(intr.number_less_than_1000 > (640*480/5)) // &&  ((intr.depth_max - intr.depth_min) > 1000))

				{
					new_min_depth = 0.8; //0.5;
				}
				else if( less_then_1500 > (640*480/5))
				{
					new_min_depth = 1.25;
				}
				else if( less_then_2000 > (640*480/5))
				{
					new_min_depth = 1.75;
				}
				else if( less_then_2500 > (640*480/5))
				{
					new_min_depth = 2.25;
				}
				else if( less_then_3000 > (640*480/5))
				{
					new_min_depth = 2.75;
				}
				else if( less_then_3500 > (640*480/5))
				{
					new_min_depth = 3.25;
					disable_weights = 1;
				}
				else
				{
					new_min_depth = 3.25;
					disable_weights = 1;
				}

				//if(depth < 0.8)
					//depth = 0.8;
				if(!disable_weights)
				{
					//if(intr.depth_min != 0)
						//mind = ((float)intr.depth_min)/1000;
					mind = new_min_depth;

					//if(intr.depth_max != 0)
						maxd = ((float)max_depth);

					float temp_max_sqr = ((mind *  mind * maxd * maxd * 15/16)/ (mind*mind - maxd*maxd/16));

					step = ((1/(mind*mind)) - (1/(temp_max_sqr)));

					weight =  (((1/(depth*depth)) - (1/(temp_max_sqr))) / step);
					//weight = weight * 64;
					//weight = fabs(sqrt(weight));

				}
				else
				// Not enough point near the camera to apply weighted ICP (i.e., without big error in measurements)
				// Switch to un-weighted ICP
				{
					weight = 1;
				}

#if RANGA_MODIFICATION_ORIENTATION
				//if(intr.number_less_than > (640*480/5))//((intr.depth_max - intr.depth_min) > 500))
				{
					float3 rayvector;
					rayvecto .x = x - intr.cx;
					rayvector.y = y - intr.cy;
					rayvector.z = (intr.fx + intr.fy)/2;

					float norm_value = norm(rayvector);

					float3 normalvector;
					float weight1 = 0.0f;
					normalvector.x = nmap_curr.ptr(y ) [x];
					normalvector.y = nmap_curr.ptr(y + rows) [x];
					normalvector.z = nmap_curr.ptr(y + 2 * rows) [x];

					float norm_value1 = norm(normalvector);

					weight1 = abs(dot(rayvector, normalvector))/(norm_value * norm_value1);

					if(weight1 > 0.6 && weight1 <= 1.0)
					{
						weight1 = (weight1 - 0.5)/ 0.5;
					}
					else if(weight1 > 1)
					{
						// This should not be reached
						weight1 = 0;
					}
					else
						weight1 = 1;

					weight = weight * weight1;

					//weight = fabs(sqrt(weight));
				}

				
#endif
				//weight = weight * 4;
				weight = fabs(sqrt(weight));
				//if(weight < 0.25)
					//weight = 0.25;
#else
				step = ((1/(min_depth)) - (1/(max_depth)));
				weight =  (((1/(depth)) - (1/(max_depth))) / step);
				weight = fabs(sqrt(weight));
#endif
			}
			else if(depth > max_depth) // || depth < min_depth)  // Minimum depth is removed as I found a case where in minimum depth is less than 0.4 m 
			// 0.8 is the minimum valid value for the kinect V1 sensor in default mode
			// 4 is the maximum valid value for kinect V1 sensor
			// http://msdn.microsoft.com/en-us/library/hh973078.aspx
			{
				weight = 0.25;
			}
			else
			{
				// As it should be square root of the actual weight
				weight = 1; //8;
			}

		  temp_1 = cross (s, n);
		  temp_2 = n;

		  temp_1.x = temp_1.x * weight ;
		  temp_1.y = temp_1.y * weight ;
		  temp_1.z = temp_1.z * weight ;

		  temp_2.x = n.x * weight;
		  temp_2.y = n.y * weight;
		  temp_2.z = n.z * weight;

#if 1
          *(float3*)&row[0] = temp_1;
          *(float3*)&row[3] = temp_2;

          row[6] = weight * dot (n, d - s);
#else
         *(float3*)&row[0] = cross (s, n);
          *(float3*)&row[3] = n;

          row[6] = dot (n, d - s);
#endif
        }
        else
          row[0] = row[1] = row[2] = row[3] = row[4] = row[5] = row[6] = 0.f;

        __shared__ float_type smem[CTA_SIZE];
        int tid = Block::flattenedThreadId ();

        int shift = 0;
        for (int i = 0; i < 6; ++i)        //rows
        {
          #pragma unroll
          for (int j = i; j < 7; ++j)          // cols + b
          {
            __syncthreads ();
            smem[tid] = row[i] * row[j];
            __syncthreads ();

            reduce<CTA_SIZE>(smem);

            if (tid == 0)
              gbuf.ptr (shift++)[blockIdx.x + gridDim.x * blockIdx.y] = smem[0];
          }
        }
      }
    };

    __global__ void
    combinedKernel (const Combined cs) 
    {
      cs ();
    }

    struct TranformReduction
    {
      enum
      {
        CTA_SIZE = 512,
        STRIDE = CTA_SIZE,

        B = 6, COLS = 6, ROWS = 6, DIAG = 6,
        UPPER_DIAG_MAT = (COLS * ROWS - DIAG) / 2 + DIAG,
        TOTAL = UPPER_DIAG_MAT + B,

        GRID_X = TOTAL
      };

      PtrStep<float_type> gbuf;
      int length;
      mutable float_type* output;

      __device__ __forceinline__ void
      operator () () const
      {
        const float_type *beg = gbuf.ptr (blockIdx.x);
        const float_type *end = beg + length;

        int tid = threadIdx.x;

        float_type sum = 0.f;
        for (const float_type *t = beg + tid; t < end; t += STRIDE)
          sum += *t;

        __shared__ float_type smem[CTA_SIZE];

        smem[tid] = sum;
        __syncthreads ();

		reduce<CTA_SIZE>(smem);

        if (tid == 0)
          output[blockIdx.x] = smem[0];
      }
    };

    __global__ void
    TransformEstimatorKernel2 (const TranformReduction tr) 
    {
      tr ();
    }
  }
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::estimateCombined (const Mat33& Rcurr, const float3& tcurr, 
                               const MapArr& vmap_curr, const MapArr& nmap_curr, 
                               const Mat33& Rprev_inv, const float3& tprev, const Intr& intr,
                               const MapArr& vmap_g_prev, const MapArr& nmap_g_prev, 
                               float distThres, float angleThres,
                               DeviceArray2D<float_type>& gbuf, DeviceArray<float_type>& mbuf, 
                               float_type* matrixA_host, float_type* vectorB_host)
{
  int cols = vmap_curr.cols ();
  int rows = vmap_curr.rows () / 3;

  Combined cs;

  cs.Rcurr = Rcurr;
  cs.tcurr = tcurr;

  cs.vmap_curr = vmap_curr;
  cs.nmap_curr = nmap_curr;

  cs.Rprev_inv = Rprev_inv;
  cs.tprev = tprev;

  cs.intr = intr;

  cs.vmap_g_prev = vmap_g_prev;
  cs.nmap_g_prev = nmap_g_prev;

  cs.distThres = distThres;
  cs.angleThres = angleThres;

  cs.cols = cols;
  cs.rows = rows;

//////////////////////////////

  dim3 block (Combined::CTA_SIZE_X, Combined::CTA_SIZE_Y);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  mbuf.create (TranformReduction::TOTAL);
  if (gbuf.rows () != TranformReduction::TOTAL || gbuf.cols () < (int)(grid.x * grid.y))
    gbuf.create (TranformReduction::TOTAL, grid.x * grid.y);

  cs.gbuf = gbuf;

  combinedKernel<<<grid, block>>>(cs);
  cudaSafeCall ( hipGetLastError () );
  //cudaSafeCall(hipDeviceSynchronize());

  //printFuncAttrib(combinedKernel);

  TranformReduction tr;
  tr.gbuf = gbuf;
  tr.length = grid.x * grid.y;
  tr.output = mbuf;

  TransformEstimatorKernel2<<<TranformReduction::TOTAL, TranformReduction::CTA_SIZE>>>(tr);
  cudaSafeCall (hipGetLastError ());
  cudaSafeCall (hipDeviceSynchronize ());

  float_type host_data[TranformReduction::TOTAL];
  mbuf.download (host_data);

  int shift = 0;
  for (int i = 0; i < 6; ++i)  //rows
    for (int j = i; j < 7; ++j)    // cols + b
    {
      float_type value = host_data[shift++];
      if (j == 6)       // vector b
        vectorB_host[i] = value;
      else
        matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
    }
}
