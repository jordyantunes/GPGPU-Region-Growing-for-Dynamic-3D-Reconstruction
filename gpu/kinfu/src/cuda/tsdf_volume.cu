#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"

using namespace pcl::device;

namespace pcl
{
  namespace device
  {
    template<typename T>
    __global__ void
    initializeVolume (PtrStep<T> volume)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;
      
      
      if (x < VOLUME_X && y < VOLUME_Y)
      {
          T *pos = volume.ptr(y) + x;
          int z_step = VOLUME_Y * volume.step / sizeof(*pos);

#pragma unroll
          for(int z = 0; z < VOLUME_Z; ++z, pos+=z_step)
             pack_tsdf (0.f, 0, *pos);
      }
    }   
  }
}

void
pcl::device::initVolume (PtrStep<short2> volume)
{
  dim3 block (32, 16);
  dim3 grid (1, 1, 1);
  grid.x = divUp (VOLUME_X, block.x);      
  grid.y = divUp (VOLUME_Y, block.y);

  initializeVolume<<<grid, block>>>(volume);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

namespace pcl
{
  namespace device
  {
    struct Tsdf
    {
      enum
      {
        CTA_SIZE_X = 32, CTA_SIZE_Y = 8,
        MAX_WEIGHT = 1 << 7
      };

      mutable PtrStep<short2> volume;
      float3 cell_size;

      Intr intr;

      Mat33 Rcurr_inv;
      float3 tcurr;

      PtrStepSz<ushort> depth_raw; //depth in mm

      float tranc_dist_mm;

      __device__ __forceinline__ float3
      getVoxelGCoo (int x, int y, int z) const
      {
        float3 coo = make_float3 (x, y, z);
        coo += 0.5f;         //shift to cell center;

        coo.x *= cell_size.x;
        coo.y *= cell_size.y;
        coo.z *= cell_size.z;

        return coo;
      }

      __device__ __forceinline__ void
      operator () () const
      {
        int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
        int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

        if (x >= VOLUME_X || y >= VOLUME_Y)
          return;

        short2 *pos = volume.ptr (y) + x;
        int elem_step = volume.step * VOLUME_Y / sizeof(*pos);

        for (int z = 0; z < VOLUME_Z; ++z, pos += elem_step)
        {
          float3 v_g = getVoxelGCoo (x, y, z);            //3 // p

          //tranform to curr cam coo space
          float3 v = Rcurr_inv * (v_g - tcurr);           //4

          int2 coo;           //project to current cam
          coo.x = __float2int_rn (v.x * intr.fx / v.z + intr.cx);
          coo.y = __float2int_rn (v.y * intr.fy / v.z + intr.cy);

          if (v.z > 0 && coo.x >= 0 && coo.y >= 0 && coo.x < depth_raw.cols && coo.y < depth_raw.rows)           //6
          {
            int Dp = depth_raw.ptr (coo.y)[coo.x];

            if (Dp != 0)
            {
              float xl = (coo.x - intr.cx) / intr.fx;
              float yl = (coo.y - intr.cy) / intr.fy;
              float lambda_inv = rsqrtf (xl * xl + yl * yl + 1);

              float sdf = 1000 * norm (tcurr - v_g) * lambda_inv - Dp; //mm

              sdf *= (-1);

              if (sdf >= -tranc_dist_mm)
              {
                float tsdf = fmin (1.f, sdf / tranc_dist_mm);

                int weight_prev;
                float tsdf_prev;

                //read and unpack
                unpack_tsdf (*pos, tsdf_prev, weight_prev);

                const int Wrk = 1;

                float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                int weight_new = min (weight_prev + Wrk, MAX_WEIGHT);

                pack_tsdf (tsdf_new, weight_new, *pos);
              }
            }
          }
        }
      }
    };

    __global__ void
    integrateTsdfKernel (const Tsdf tsdf) {
      tsdf ();
    }

    __global__ void
    tsdf2 (PtrStep<short2> volume, const float tranc_dist_mm, const Mat33 Rcurr_inv, float3 tcurr,
           const Intr intr, const PtrStepSz<ushort> depth_raw, const float3 cell_size)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      short2 *pos = volume.ptr (y) + x;
      int elem_step = volume.step * VOLUME_Y / sizeof(short2);

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_x = Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z;
      float v_y = Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z;
      float v_z = Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z;

//#pragma unroll
      for (int z = 0; z < VOLUME_Z; ++z)
      {
        float3 vr;
        vr.x = v_g_x;
        vr.y = v_g_y;
        vr.z = (v_g_z + z * cell_size.z);

        float3 v;
        v.x = v_x + Rcurr_inv.data[0].z * z * cell_size.z;
        v.y = v_y + Rcurr_inv.data[1].z * z * cell_size.z;
        v.z = v_z + Rcurr_inv.data[2].z * z * cell_size.z;

        int2 coo;         //project to current cam
        coo.x = __float2int_rn (v.x * intr.fx / v.z + intr.cx);
        coo.y = __float2int_rn (v.y * intr.fy / v.z + intr.cy);


        if (v.z > 0 && coo.x >= 0 && coo.y >= 0 && coo.x < depth_raw.cols && coo.y < depth_raw.rows)         //6
        {
          int Dp = depth_raw.ptr (coo.y)[coo.x]; //mm

          if (Dp != 0)
          {
            float xl = (coo.x - intr.cx) / intr.fx;
            float yl = (coo.y - intr.cy) / intr.fy;
            float lambda_inv = rsqrtf (xl * xl + yl * yl + 1);

            float sdf = Dp - norm (vr) * lambda_inv * 1000; //mm


            if (sdf >= -tranc_dist_mm)
            {
              float tsdf = fmin (1.f, sdf / tranc_dist_mm);

              int weight_prev;
              float tsdf_prev;

              //read and unpack
              unpack_tsdf (*pos, tsdf_prev, weight_prev);

              const int Wrk = 1;

              float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
              int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

              pack_tsdf (tsdf_new, weight_new, *pos);
            }
          }
        }
        pos += elem_step;
      }       /* for(int z = 0; z < VOLUME_Z; ++z) */
    }      /* __global__ */
  }
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::integrateTsdfVolume (const PtrStepSz<ushort>& depth_raw, const Intr& intr, const float3& volume_size,
                                  const Mat33& Rcurr_inv, const float3& tcurr, float tranc_dist, 
                                  PtrStep<short2> volume)
{
  Tsdf tsdf;

  tsdf.volume = volume;  
  tsdf.cell_size.x = volume_size.x / VOLUME_X;
  tsdf.cell_size.y = volume_size.y / VOLUME_Y;
  tsdf.cell_size.z = volume_size.z / VOLUME_Z;
  
  tsdf.intr = intr;

  tsdf.Rcurr_inv = Rcurr_inv;
  tsdf.tcurr = tcurr;
  tsdf.depth_raw = depth_raw;

  tsdf.tranc_dist_mm = tranc_dist*1000; //mm

  dim3 block (Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
  dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

#if 0
   //tsdf2<<<grid, block>>>(volume, tranc_dist, Rcurr_inv, tcurr, intr, depth_raw, tsdf.cell_size);
   integrateTsdfKernel<<<grid, block>>>(tsdf);
#endif
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}


namespace pcl
{
  namespace device
  {
    __global__ void
    scaleDepth (const PtrStepSz<ushort> depth, PtrStep<float> scaled, const Intr intr)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= depth.cols || y >= depth.rows)
        return;

      int Dp = depth.ptr (y)[x];

      float xl = (x - intr.cx) / intr.fx;
      float yl = (y - intr.cy) / intr.fy;
      float lambda = sqrtf (xl * xl + yl * yl + 1);

      scaled.ptr (y)[x] = Dp * lambda/1000.f; //meters
    }

    __global__ void
    tsdf23 (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
            const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= VOLUME_X || y >= VOLUME_Y)
        return;

      float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
      float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
      float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

      float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

      float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
      float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
      float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

      float z_scaled = 0;

      float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
      float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

      float tranc_dist_inv = 1.0f / tranc_dist;

      short2* pos = volume.ptr (y) + x;
      int elem_step = volume.step * VOLUME_Y / sizeof(short2);

//#pragma unroll
      for (int z = 0; z < VOLUME_Z;
           ++z,
           v_g_z += cell_size.z,
           z_scaled += cell_size.z,
           v_x += Rcurr_inv_0_z_scaled,
           v_y += Rcurr_inv_1_z_scaled,
           pos += elem_step)
      {
        float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
        if (inv_z < 0)
            continue;

        // project to current cam
        int2 coo =
        {
          __float2int_rn (v_x * inv_z + intr.cx),
          __float2int_rn (v_y * inv_z + intr.cy)
        };

        if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
        {
          float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

          float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

          if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
          {
            float tsdf = fmin (1.0f, sdf * tranc_dist_inv);

            //read and unpack
            float tsdf_prev;
            int weight_prev;
            unpack_tsdf (*pos, tsdf_prev, weight_prev);

            const int Wrk = 1;

            float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
            int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

            pack_tsdf (tsdf_new, weight_new, *pos);
          }
        }
      }       // for(int z = 0; z < VOLUME_Z; ++z)
    }      // __global__

    __global__ void
    tsdf23normal_hack (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
                  const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size)
    {
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y;

        if (x >= VOLUME_X || y >= VOLUME_Y)
            return;

        const float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
        const float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
        float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

        float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

        float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
        float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
        float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

        float z_scaled = 0;

        float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
        float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

        float tranc_dist_inv = 1.0f / tranc_dist;

        short2* pos = volume.ptr (y) + x;
        int elem_step = volume.step * VOLUME_Y / sizeof(short2);

        //#pragma unroll
        for (int z = 0; z < VOLUME_Z;
            ++z,
            v_g_z += cell_size.z,
            z_scaled += cell_size.z,
            v_x += Rcurr_inv_0_z_scaled,
            v_y += Rcurr_inv_1_z_scaled,
            pos += elem_step)
        {
            float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
            if (inv_z < 0)
                continue;

            // project to current cam
            int2 coo =
            {
                __float2int_rn (v_x * inv_z + intr.cx),
                __float2int_rn (v_y * inv_z + intr.cy)
            };

            if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)         //6
            {
                float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

                float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

                if (Dp_scaled != 0 && sdf >= -tranc_dist) //meters
                {
                    float tsdf = fmin (1.0f, sdf * tranc_dist_inv);                                              

                    bool integrate = true;
                    if ((x > 0 &&  x < VOLUME_X-2) && (y > 0 && y < VOLUME_Y-2) && (z > 0 && z < VOLUME_Z-2))
                    {
                        const float qnan = numeric_limits<float>::quiet_NaN();
                        float3 normal = make_float3(qnan, qnan, qnan);

                        float Fn, Fp;
                        int Wn = 0, Wp = 0;
                        unpack_tsdf (*(pos + elem_step), Fn, Wn);
                        unpack_tsdf (*(pos - elem_step), Fp, Wp);

                        if (Wn > 16 && Wp > 16) 
                            normal.z = (Fn - Fp)/cell_size.z;

                        unpack_tsdf (*(pos + volume.step/sizeof(short2) ), Fn, Wn);
                        unpack_tsdf (*(pos - volume.step/sizeof(short2) ), Fp, Wp);

                        if (Wn > 16 && Wp > 16) 
                            normal.y = (Fn - Fp)/cell_size.y;

                        unpack_tsdf (*(pos + 1), Fn, Wn);
                        unpack_tsdf (*(pos - 1), Fp, Wp);

                        if (Wn > 16 && Wp > 16) 
                            normal.x = (Fn - Fp)/cell_size.x;

                        if (normal.x != qnan && normal.y != qnan && normal.z != qnan)
                        {
                            float norm2 = dot(normal, normal);
                            if (norm2 >= 1e-10)
                            {
                                normal *= rsqrt(norm2);

                                float nt = v_g_x * normal.x + v_g_y * normal.y + v_g_z * normal.z;
                                float cosine = nt * rsqrt(v_g_x * v_g_x + v_g_y * v_g_y + v_g_z * v_g_z);

                                if (cosine < 0.5)
                                    integrate = false;
                            }
                        }
                    }

                    if (integrate)
                    {
                        //read and unpack
                        float tsdf_prev;
                        int weight_prev;
                        unpack_tsdf (*pos, tsdf_prev, weight_prev);

                        const int Wrk = 1;

                        float tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
                        int weight_new = min (weight_prev + Wrk, Tsdf::MAX_WEIGHT);

                        pack_tsdf (tsdf_new, weight_new, *pos);
                    }
                }
            }
        }       // for(int z = 0; z < VOLUME_Z; ++z)
    }      // __global__

	//__global__ void
	//	tsdf23_weight_aware (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
	//	const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size
	//	, const ConfigDASS cfg, const PtrStepSz<unsigned char> indicator
	//	)

    __global__ void
		tsdf23_weight_aware (const PtrStepSz<float> depthScaled, PtrStep<short2> volume,
		const float tranc_dist, const Mat33 Rcurr_inv, const float3 tcurr, const Intr intr, const float3 cell_size
		, const ConfigDASS cfg)
	{

		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if (x >= VOLUME_X || y >= VOLUME_Y)
			return;

		float v_g_x = (x + 0.5f) * cell_size.x - tcurr.x;
		float v_g_y = (y + 0.5f) * cell_size.y - tcurr.y;
		float v_g_z = (0 + 0.5f) * cell_size.z - tcurr.z;

		float v_g_part_norm = v_g_x * v_g_x + v_g_y * v_g_y;

		float v_x = (Rcurr_inv.data[0].x * v_g_x + Rcurr_inv.data[0].y * v_g_y + Rcurr_inv.data[0].z * v_g_z) * intr.fx;
		float v_y = (Rcurr_inv.data[1].x * v_g_x + Rcurr_inv.data[1].y * v_g_y + Rcurr_inv.data[1].z * v_g_z) * intr.fy;
		float v_z = (Rcurr_inv.data[2].x * v_g_x + Rcurr_inv.data[2].y * v_g_y + Rcurr_inv.data[2].z * v_g_z);

		float z_scaled = 0;

		float Rcurr_inv_0_z_scaled = Rcurr_inv.data[0].z * cell_size.z * intr.fx;
		float Rcurr_inv_1_z_scaled = Rcurr_inv.data[1].z * cell_size.z * intr.fy;

		float tranc_dist_inv = 1.0f / tranc_dist;

		short2* pos = volume.ptr (y) + x;
		int elem_step = volume.step * VOLUME_Y / sizeof(short2);

		//#pragma unroll
		for (int z = 0; z < VOLUME_Z;
			++z,
			v_g_z += cell_size.z,
			z_scaled += cell_size.z,
			v_x += Rcurr_inv_0_z_scaled,
			v_y += Rcurr_inv_1_z_scaled,
			pos += elem_step)
		{
			float inv_z = 1.0f / (v_z + Rcurr_inv.data[2].z * z_scaled);
			if (inv_z < 0)
				continue;

			// project to current cam
			int2 coo =
			{
				__float2int_rn (v_x * inv_z + intr.cx),
				__float2int_rn (v_y * inv_z + intr.cy)
			};

			if (coo.x >= 0 && coo.y >= 0 && coo.x < depthScaled.cols && coo.y < depthScaled.rows)
			{
				float Dp_scaled = depthScaled.ptr (coo.y)[coo.x]; //meters

				float sdf = Dp_scaled - sqrtf (v_g_z * v_g_z + v_g_part_norm);

				if (Dp_scaled != 0 && sdf >= -tranc_dist ) //meters
				{
					float tsdf = fmin (1.0f, sdf * tranc_dist_inv);

					//read and unpack
					float tsdf_prev;
					int weight_prev;
					unpack_tsdf (*pos, tsdf_prev, weight_prev);
					// ********************************************************************
					// Various weight definitions and updating strategies:
					//    DA:   weight_mode = 5,   update_mode = 7
					//    DASS: weight_mode = 5,   update_mode = 8
					// ********************************************************************
					int Wrk = 1;
					float step = 1.0f;

					// to choose weight definition based on 'weight_mode'
					switch ( cfg.weight_mode )
					{
					default: // UNKNOWN mode !!!
					case 1: // constant weight = 1 (KinFu default) multiplied by scale
						Wrk = (int)(1 * cfg.weight_scale);
						break;

					case 2: // weight ~ distance
					case 3:
					case 4:					
					case 5: // weight ~ 1/distance^2
						// checks the limits
						if ( Dp_scaled >= cfg.minimum_distance && Dp_scaled <= cfg.maximum_distance )
						{
							step = ((1/(cfg.minimum_distance*cfg.minimum_distance)) - (1/(cfg.maximum_distance*cfg.maximum_distance))) / cfg.weight_bins;
							Wrk = (int)( (((1/(Dp_scaled*Dp_scaled)) - (1/(cfg.maximum_distance*cfg.maximum_distance))) / step) * cfg.weight_scale);
						}
						else
						{
							Wrk = 0;
						}
						break;

					

					//  case 6: // weight ~ Indicator 
					//	Wrk = indicator.ptr (coo.y)[coo.x] ;
					//	// Wrk = 0; 
					//	// if (Wrk == 51)
					//	// printf("%d %d %d\n", Wrk, coo.x, coo.y);
					//	break;
					}

					float tsdf_new = 0;
					float difference_by_stable_value(0);
					int new_weight_temporary = 0;

					int tmp = 0;
					unsigned char weight_8bit = 0;
					unsigned char last_weight_8bit = 0;

					// to choose weight-update strategy based on 'weight_update'
					switch ( cfg.weight_update )
					{
						//case 0: // NEVER UPDATE
						//    be_update = false;
						//    break;

					default: // UNKNOWN update strategy !!!
					case 1: // accumulating (tsdf default)
						tsdf_new = (tsdf_prev * weight_prev + Wrk * tsdf) / (weight_prev + Wrk);
						new_weight_temporary = weight_prev + Wrk;
						break;

					case 2: 
					case 3:
					case 4:
					case 5:
					case 6:
						break;

					case 7: // maximizing: updates UNTIL graeter values are available (in a range)

						// weight_prev (HL 16-bit)   =   last_weight_8bit (H 8-bit)   +   weight_8bit (L 8-bit)
						weight_8bit = weight_prev;
						last_weight_8bit = weight_prev >> 8;

						if ( Wrk >= last_weight_8bit*cfg.weight_range )
						{
							// same as original
							tsdf_new = (tsdf_prev * weight_8bit + Wrk * tsdf) / (weight_8bit + Wrk);

							// updates the last_weight_8bit
							new_weight_temporary = min(255, weight_8bit + Wrk); // 8-bit so maximum =255

							if ( Wrk > last_weight_8bit ) last_weight_8bit = Wrk;

							new_weight_temporary += last_weight_8bit << 8;
						}
						else
						{
							tsdf_new = tsdf_prev;
							new_weight_temporary = weight_prev;
						}
						break;

					case 8: // maximizing: updates UNTIL greater values are available (in a range) Wrk=1
						// try to be more similar to the original tsdf model

						// weight_prev (HL 16-bit)   =   last_weight_8bit (H 8-bit)   +   weight_8bit (L 8-bit)
						weight_8bit = weight_prev;
						last_weight_8bit = weight_prev >> 8;

						if ( Wrk >= last_weight_8bit*cfg.weight_range )
						{
							// same as original
							tsdf_new = (tsdf_prev * weight_8bit + 1 * tsdf) / (weight_8bit + 1);

							// updates the last_weight_8bit
							new_weight_temporary = min(cfg.maximum_weight, weight_8bit + 1);

							if ( Wrk > last_weight_8bit ) last_weight_8bit = Wrk;

							new_weight_temporary += last_weight_8bit << 8;
						}
						else
						{
							tsdf_new = tsdf_prev;
							new_weight_temporary = weight_prev;
						}
						break;
					}

					// checks the maximum weight limit
					int weight_new = 0;
					if ( cfg.weight_update == 7 || cfg.weight_update == 8 ) weight_new = new_weight_temporary;
					else weight_new = min (new_weight_temporary, cfg.maximum_weight );

					// ********************************************************************
					// ********************************************************************
					if ( cfg.weight_update > 0 ) pack_tsdf (tsdf_new, weight_new, *pos);
				}
			}
		}       // for(int z = 0; z < VOLUME_Z; ++z)
	}      // __global__

  }
}



//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::integrateTsdfVolume (const PtrStepSz<ushort>& depth, const Intr& intr,
                                  const float3& volume_size, const Mat33& Rcurr_inv, const float3& tcurr, 
                                  float tranc_dist,
                                  PtrStep<short2> volume, DeviceArray2D<float>& depthScaled)
{
  // Maximum value the weight can take = 8
  // Maximum value total weight can take = 255
  // Mode = 5 and Update = 8
  // Bins = 1 ??
  // Minimum Distance = 0.8 m
  // Maximum Distance = 10 m
  // Ratio of maximum weight for updation = 80%
  ConfigDASS cfg(5, 8, 1, 255, 0.8, 4.0, 0.80, 255);
  // ConfigDASS cfg(5, 8, 1, 255, 0.4, 6.0, 0.80, 255);
  depthScaled.create (depth.rows, depth.cols);

  dim3 block_scale (32, 8);
  dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));

  //scales depth along ray and converts mm -> meters. 
  scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
  cudaSafeCall ( hipGetLastError () );

  float3 cell_size;
  cell_size.x = volume_size.x / VOLUME_X;
  cell_size.y = volume_size.y / VOLUME_Y;
  cell_size.z = volume_size.z / VOLUME_Z;

  //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
  dim3 block (16, 16);
  dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));

  PtrStepSz<unsigned char> indicator;
  //tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);
  //tsdf23_weight_aware<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size, cfg, indicator);
  tsdf23_weight_aware<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size, cfg);

  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

//// modified version for indicator map
//void
//pcl::device::integrateTsdfVolume (const PtrStepSz<ushort>& depth, const Intr& intr,
//                                  const float3& volume_size, const Mat33& Rcurr_inv, const float3& tcurr, 
//                                  float tranc_dist,
//                                  PtrStep<short2> volume, DeviceArray2D<float>& depthScaled, const PtrStepSz<unsigned char>& indicator)
//{
//  // Maximum value the weight can take = 8
//  // Maximum value total weight can take = 255
//  // Mode = 5 and Update = 8
//  // Bins = 1 ??
//  // Minimum Distance = 0.8 m
//  // Maximum Distance = 10 m
//  // Ratio of maximum weight for updation = 80%
//
// 
//
//  ConfigDASS cfg(6, 8, 1, 255, 0.8, 4.0, 0.80, 255);
//  // ConfigDASS cfg(5, 8, 1, 255, 0.4, 6.0, 0.80, 255);
//  depthScaled.create (depth.rows, depth.cols);
//
//  dim3 block_scale (32, 8);
//  dim3 grid_scale (divUp (depth.cols, block_scale.x), divUp (depth.rows, block_scale.y));
//
//  //scales depth along ray and converts mm -> meters. 
//  scaleDepth<<<grid_scale, block_scale>>>(depth, depthScaled, intr);
//  cudaSafeCall ( hipGetLastError () );
//
//  /*
//  int a = 200;
//  int b = 200;
//
//  int test = depthScaled.ptr(a)[b];
//  printf("%i\n",test);
//  */
//
//  float3 cell_size;
//  cell_size.x = volume_size.x / VOLUME_X;
//  cell_size.y = volume_size.y / VOLUME_Y;
//  cell_size.z = volume_size.z / VOLUME_Z;
//
//  //dim3 block(Tsdf::CTA_SIZE_X, Tsdf::CTA_SIZE_Y);
//  dim3 block (16, 16);
//  dim3 grid (divUp (VOLUME_X, block.x), divUp (VOLUME_Y, block.y));
//
//  //tsdf23<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size);
//  tsdf23_weight_aware<<<grid, block>>>(depthScaled, volume, tranc_dist, Rcurr_inv, tcurr, intr, cell_size, cfg, indicator);
//
//  cudaSafeCall ( hipGetLastError () );
//  cudaSafeCall (hipDeviceSynchronize ());
//}